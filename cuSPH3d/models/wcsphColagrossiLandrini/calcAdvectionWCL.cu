#include "hip/hip_runtime.h"
/*
* calcAdvectionWCL.cu
*
*  Created on: 16-12-2013
*      Author: Kamil Szewc (kamil.szewc@gmail.com)
*/
#include "../../sph.h"

__global__ void calcAdvectionWCL(Particle *p, Parameters *par)
{
	uint tid = threadIdx.x + blockIdx.x * blockDim.x;
	while (tid < par->N) {
		p[tid].vel.x += par->DT * ((1.0 - par->XSPH)*p[tid].rh_vel.x + par->XSPH*p[tid].rh_pos.x + p[tid].st.x + par->G_X);
		p[tid].vel.y += par->DT * ((1.0 - par->XSPH)*p[tid].rh_vel.y + par->XSPH*p[tid].rh_pos.y + p[tid].st.y + par->G_Y);
        p[tid].vel.z += par->DT * ((1.0 - par->XSPH)*p[tid].rh_vel.z + par->XSPH*p[tid].rh_pos.z + p[tid].st.z + par->G_Z);
        p[tid].pos.x += par->DT * p[tid].vel.x;
		p[tid].pos.y += par->DT * p[tid].vel.y;
        p[tid].pos.z += par->DT * p[tid].vel.z;
		p[tid].d += par->DT * p[tid].rh_d;

		if (par->T_BOUNDARY_PERIODICITY == 0) //X,Y
		{
			if (p[tid].pos.x > par->XCV)
			{
				p[tid].vel.x = -p[tid].vel.x;
				p[tid].pos.x = 2.0 * par->XCV - p[tid].pos.x;
			}
			if (p[tid].pos.x <= 0.0)
			{
				p[tid].vel.x = -p[tid].vel.x;
				p[tid].pos.x = -p[tid].pos.x;
			}

			if (p[tid].pos.y > par->YCV)
			{
				p[tid].vel.y = -p[tid].vel.y;
				p[tid].pos.y = 2.0 * par->YCV - p[tid].pos.y;
			}
			if (p[tid].pos.y <= 0.0)
			{
				p[tid].vel.y = -p[tid].vel.y;
				p[tid].pos.y = -p[tid].pos.y;
			}
		}
		else
		{
			if (p[tid].pos.x > par->XCV) p[tid].pos.x -= par->XCV;
			if (p[tid].pos.x <= 0.0)    p[tid].pos.x += par->XCV;
			if (p[tid].pos.y > par->YCV) p[tid].pos.y -= par->YCV;
			if (p[tid].pos.y <= 0.0)    p[tid].pos.y += par->YCV;
		}

		if ((par->T_BOUNDARY_PERIODICITY == 0) || (par->T_BOUNDARY_PERIODICITY == 2))  //Z
		{

			if (p[tid].pos.z > par->ZCV)
			{
				p[tid].vel.z = -p[tid].vel.z;
				p[tid].pos.z = 2.0 * par->ZCV - p[tid].pos.z;
			}
			if (p[tid].pos.z <= 0.0)
			{
				p[tid].vel.z = -p[tid].vel.z;
				p[tid].pos.z = -p[tid].pos.z;
			}
		}
		else
		{
			if (p[tid].pos.z > par->ZCV) p[tid].pos.z -= par->ZCV;
			if (p[tid].pos.z <= 0.0)    p[tid].pos.z += par->ZCV;
		}

		tid += blockDim.x * gridDim.x;
	}

}
