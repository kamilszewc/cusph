#include "hip/hip_runtime.h"
/*
* calcInteractionWCL.cu
*
*  Created on: 16-12-2013
*      Author: Kamil Szewc (kamil.szewc@gmail.com)
*/

#include "../../sph.h"
#include "../../hlp.h"
#include "../../methods/kernels.cuh"
#include "../../methods/interactions.cuh"

__device__ static real4 interaction(uint i, uint j, real3 dpos, real3 dvel, real3 dvelSlip, Particle *p, Parameters *par)
{
	real r = sqrt(pow2(dpos.x) + pow2(dpos.y) + pow2(dpos.z));
	real q = r * par->I_H;
	if (q < 2.0) 
	{
		real gkx = grad_of_kern(dpos.x, q, par->GKNORM);
		real gky = grad_of_kern(dpos.y, q, par->GKNORM);
		real gkz = grad_of_kern(dpos.z, q, par->GKNORM);

		real pres = (p[i].p + p[j].p) / (p[i].d * p[j].d);
		if ((par->T_INTERFACE_CORRECTION == 1) && (p[i].c != p[j].c)) pres += par->INTERFACE_CORRECTION * fabs(p[i].p + p[j].p) / (p[i].d*p[j].d);

		real visc = 8.0*(p[i].nu + p[j].nu) * (dvel.x*dpos.x + dvel.y*dpos.y + dvel.z*dpos.z) / ((r*r+0.01*pow2(par->H)) * (p[i].d * p[j].d));

		real dens = dvelSlip.x*gkx + dvelSlip.y*gky + dvelSlip.z*gkz;

		return MAKE_REAL4(p[j].m*(visc - pres)*gkx, p[j].m*(visc - pres)*gky, p[j].m*(visc - pres)*gkz, dens*p[j].m / p[j].d);
	}
	else {
		return MAKE_REAL4(0.0, 0.0, 0.0, 0.0);
	}
}



__global__ void calcInteractionWCL(Particle *p,
								   uint *gridParticleIndex,
								   uint *cellStart,
								   uint *cellEnd,
								   Parameters *par)
{
	uint index = threadIdx.x + blockIdx.x*blockDim.x;
	if (index < par->N) {

		register real4 result = MAKE_REAL4(0.0, 0.0,0.0,0.0);
		#include "../../methods/interactions/interactions_3NegativeOnWallNoSlip_1PositiveOnWallFreeSlip.cuh"

		p[index].rh_vel.x = result.x;
		p[index].rh_vel.y = result.y;
		p[index].rh_vel.z = result.z;
		p[index].rh_d = p[index].d * result.w;
	}
}
