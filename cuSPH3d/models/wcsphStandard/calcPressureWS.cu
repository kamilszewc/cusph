#include "hip/hip_runtime.h"
/*
* calcPressureWS.cu
*
*  Created on: 17-12-2013
*      Author: Kamil Szewc
*/
#include "../../sph.h"
#include "../../hlp.h"

__global__ void calcPressureWS(Particle *p, Parameters *par)
{
	unsigned int tid = threadIdx.x + blockIdx.x*blockDim.x;
	while (tid < par->N) {
		p[tid].p = p[tid].b * (pow(p[tid].d / p[tid].di, p[tid].gamma) - 1.0);
		tid += blockDim.x * gridDim.x;
	}
}


