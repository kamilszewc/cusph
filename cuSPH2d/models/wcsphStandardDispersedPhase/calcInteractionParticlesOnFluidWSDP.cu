#include "hip/hip_runtime.h"
/*
* calcInteractionParticlesOnFluid.cu
*
*  Created on: 27-10-2015
*      Author: Kamil Szewc (kamil.szewc@gmail.com)
*/
#include "../../sph.h"
#include "../../hlp.h"
#include "../../methods/kernels.cuh"
#include "../../methods/calcGridHash.cuh"
#include "../../methods/calcGridPos.cuh"
#include "../../methods/calcRelPosVelNoSlip.cuh"
#include "wcsphStandardDispersedPhase.h"

__device__ static float calcK(real2 dvel, real dLiquid, real oLiquid, real oDust, real d, real visc)
{
	real u = sqrt(pow2(dvel.x) + pow2(dvel.y));

	real re = d * u / visc;

	real cd = 0.0;
	if (re > 0.0) cd = 24.0 * sqrt(1.0 + (3.0 * re / 16.0)) / re;

	return (3.0 / 4.0) * dLiquid * oLiquid * oDust * cd * u * pow(oLiquid, -2.65) / d;
}

__device__ static real6 interaction(uint j, Particle *p, uint i, Particle *pFluid, real2 dpos, real2 dvel, Parameters *par)
{
	real r = sqrt(pow2(dpos.x) + pow2(dpos.y));
	real q = r * par->I_H;

	if (q < 2.0)
	{
		real gkx = grad_of_kern(dpos.x, q, par->I_H);
		real gky = grad_of_kern(dpos.y, q, par->I_H);
		real k = kern_kwon_monaghan(q, par->I_H);
			
		real D = 0.0005;

		real K = calcK(dvel, pFluid[i].d, pFluid[i].o, p[j].o, D, pFluid[i].nu);

		real pres = p[j].m * p[j].o / p[j].d;

		real visc = 2.0 * p[j].m * K * (dvel.x*dpos.x + dvel.y*dpos.y) * k / ((pow2(r) + 0.01*pow2(par->H)) * p[j].d);
		
		real dens = dvel.x*gkx + dvel.y*gky;

		return MAKE_REAL6(pres * gkx, pres * gky, visc * dpos.x, visc * dpos.y, p[j].m * dens, 0.0);
	}
	else
	{
		return MAKE_REAL6(0.0, 0.0, 0.0, 0.0, 0.0, 0.0);
	}
}



__global__ void calcInteractionParticlesOnFluidWSDP(Particle *p,
	uint *gridParticleIndex,
	uint *cellStart,
	uint *cellEnd,
	Particle *pPDPF,
	Parameters *par)
{
	uint index = threadIdx.x + blockIdx.x*blockDim.x;
	if (index < par->N) {

		real2 pos = MAKE_REAL2(p[index].pos.x, p[index].pos.y);

		int2 gridPos = calcGridPos(pos, par);
		uint gridHash0 = calcGridHash(gridPos, par);
		real6 result = MAKE_REAL6(0.0, 0.0, 0.0, 0.0, 0.0, 0.0);

		for (int y = -1; y <= 1; y++) {
			for (int x = -1; x <= 1; x++) {
				int2 gridPos2;
				gridPos2.x = gridPos.x + x;
				gridPos2.y = gridPos.y + y;
				if ((gridPos2.x < 0) || (gridPos2.x > par->NXC - 1) || (gridPos2.y < 0) || (gridPos2.y > par->NYC - 1)) continue;

				uint gridHash = calcGridHash(gridPos2, par);
				uint startIndex = cellStart[gridHash];

				if (startIndex != 0xffffffff)
				{
					uint endIndex = cellEnd[gridHash];

					for (uint j = startIndex; j < endIndex; j++)
					{
						real2 dpos, dvel;
						real2 pos1 = p[index].pos;
						real2 pos2 = pPDPF[j].pos;
						real2 vel1 = p[index].vel;
						real2 vel2 = pPDPF[j].vel;
						calcRelPosVelNoSlip(pos1, pos2, vel1, vel2, 0, &dpos, &dvel, par);
						result += interaction(j, pPDPF, index, p, dpos, dvel, par);

						if (((gridPos.x == 0) && (gridPos2.x == 0)) || ((gridPos.x == par->NXC - 1) && (gridPos2.x == par->NXC - 1)) ||
							((gridPos.y == 0) && (gridPos2.y == 0)) || ((gridPos.y == par->NYC - 1) && (gridPos2.y == par->NYC - 1)))
						{
							if (par->T_BOUNDARY_PERIODICITY != 1)
							{
								if (gridPos.y == gridPos2.y)
								{
									if (gridPos.y == par->NYC - 1)
									{
										calcRelPosVelNoSlip(pos2, pos1, vel2, vel1, 1, &dpos, &dvel, par);
										result += interaction(j, pPDPF, index, p, dpos, dvel, par);
									}
									if (gridPos.y == 0)
									{
										calcRelPosVelNoSlip(pos2, pos1, vel2, vel1, 3, &dpos, &dvel, par);
										result += interaction(j, pPDPF, index, p, dpos, dvel, par);
									}
								}
							}
							if (par->T_BOUNDARY_PERIODICITY == 0)
							{
								if (gridPos.x == gridPos2.x)
								{
									if (gridPos2.x == 0)
									{
										calcRelPosVelNoSlip(pos2, pos1, vel2, vel1, 4, &dpos, &dvel, par);
										result += interaction(j, pPDPF, index, p, dpos, dvel, par);
									}
									if (gridPos2.x == par->NXC - 1)
									{
										calcRelPosVelNoSlip(pos2, pos1, vel2, vel1, 2, &dpos, &dvel, par);
										result += interaction(j, pPDPF, index, p, dpos, dvel, par);
									}
								}
								if ((gridPos.x == gridPos2.x) && (gridPos.y == gridPos.y))
								{
									if ((gridPos.x == 0) && (gridPos.y == 0))
									{
										calcRelPosVelNoSlip(pos2, pos1, vel2, vel1, 7, &dpos, &dvel, par);
										result += interaction(j, pPDPF, index, p, dpos, dvel, par);
									}
									if ((gridPos.x == par->NXC - 1) && (gridPos.y == 0))
									{
										calcRelPosVelNoSlip(pos2, pos1, vel2, vel1, 6, &dpos, &dvel, par);
										result += interaction(j, pPDPF, index, p, dpos, dvel, par);
									}
									if ((gridPos.x == 0) && (gridPos.y == par->NYC - 1))
									{
										calcRelPosVelNoSlip(pos2, pos1, vel2, vel1, 8, &dpos, &dvel, par);
										result += interaction(j, pPDPF, index, p, dpos, dvel, par);
									}
									if ((gridPos.x == par->NXC - 1) && (gridPos.y == par->NYC - 1))
									{
										calcRelPosVelNoSlip(pos2, pos1, vel2, vel1, 5, &dpos, &dvel, par);
										result += interaction(j, pPDPF, index, p, dpos, dvel, par);
									}
								}
							}

						}
					}

				}
			}
		}

		if ((par->T_BOUNDARY_PERIODICITY > 0) && ((gridPos.x == 0) || (gridPos.x == par->NXC - 1)))
		{
			for (int y = -1; y <= 1; y++)
			{
				int2 gridPos2;
				if (gridPos.x == 0) gridPos2.x = par->NXC - 1;
				if (gridPos.x == par->NXC - 1) gridPos2.x = 0;
				gridPos2.y = gridPos.y + y;
				if ((gridPos2.y < 0) || (gridPos2.y > par->NYC - 1)) continue;

				uint gridHash = calcGridHash(gridPos2, par);
				uint startIndex = cellStart[gridHash];

				if (startIndex != 0xffffffff)
				{
					uint endIndex = cellEnd[gridHash];

					for (uint j = startIndex; j < endIndex; j++)
					{
						real2 dpos, dvel;
						real2 pos1 = p[index].pos;
						if (gridPos.x == 0) pos1.x += par->XCV;
						if (gridPos.x == par->NXC - 1) pos1.x -= par->XCV;
						real2 pos2 = pPDPF[j].pos;
						real2 vel1 = p[index].vel;
						real2 vel2 = pPDPF[j].vel;

						calcRelPosVelNoSlip(pos1, pos2, vel1, vel2, 0, &dpos, &dvel, par);
						result += interaction(j, pPDPF, index, p, dpos, dvel, par);
					}

				}
			}
		}


		if ((par->T_BOUNDARY_PERIODICITY == 1) && ((gridPos.y == 0) || (gridPos.y == par->NYC - 1)))
		{
			for (int x = -1; x <= 1; x++)
			{
				int2 gridPos2;
				if (gridPos.y == 0) gridPos2.y = par->NYC - 1;
				if (gridPos.y == par->NYC - 1) gridPos2.y = 0;
				gridPos2.x = gridPos.x + x;
				if ((gridPos2.x < 0) || (gridPos2.x > par->NXC - 1)) continue;

				uint gridHash = calcGridHash(gridPos2, par);
				uint startIndex = cellStart[gridHash];

				if (startIndex != 0xffffffff)
				{
					uint endIndex = cellEnd[gridHash];

					for (uint j = startIndex; j < endIndex; j++)
					{
						real2 dpos, dvel;
						real2 pos1 = p[index].pos;
						if (gridPos.y == 0) pos1.y += par->YCV;
						if (gridPos.y == par->NYC - 1) pos1.y -= par->YCV;
						real2 pos2 = pPDPF[j].pos;
						real2 vel1 = p[index].vel;
						real2 vel2 = pPDPF[j].vel;

						calcRelPosVelNoSlip(pos1, pos2, vel1, vel2, 0, &dpos, &dvel, par);
						result += interaction(j, pPDPF, index, p, dpos, dvel, par);
					}

				}
			}
		}

		if (par->T_BOUNDARY_PERIODICITY == 1)
		{
			if (((gridPos.x == 0) && (gridPos.y == 0))
				|| (gridPos.x == 0) && (gridPos.y == par->NYC - 1)
				|| (gridPos.x == par->NXC - 1) && (gridPos.y == 0)
				|| (gridPos.x == par->NXC - 1) && (gridPos.y == par->NYC - 1))
			{
				int2 gridPos2;

				if (gridPos.x == 0) gridPos2.x = par->NXC - 1;
				else gridPos2.x = 0;

				if (gridPos.y == 0) gridPos2.y = par->NYC - 1;
				else gridPos2.y = 0;

				uint gridHash = calcGridHash(gridPos2, par);
				uint startIndex = cellStart[gridHash];

				if (startIndex != 0xffffffff)
				{
					uint endIndex = cellEnd[gridHash];
					for (uint j = startIndex; j < endIndex; j++)
					{
						real2 dpos, dvel;
						real2 pos1 = p[index].pos;
						if (gridPos.x == 0) pos1.x += par->XCV;
						else pos1.x -= par->XCV;
						if (gridPos.y == 0) pos1.y += par->YCV;
						else pos1.y -= par->YCV;
						real2 pos2 = pPDPF[j].pos;
						real2 vel1 = p[index].vel;
						real2 vel2 = pPDPF[j].vel;

						calcRelPosVelNoSlip(pos1, pos2, vel1, vel2, 0, &dpos, &dvel, par);
						result += interaction(j, pPDPF, index, p, dpos, dvel, par);
					}
				}
			}

		}

		if (par->T_BOUNDARY_PERIODICITY == 2)
		{
			if (((gridPos.x == 0) && (gridPos.y == 0))
				|| (gridPos.x == 0) && (gridPos.y == par->NYC - 1)
				|| (gridPos.x == par->NXC - 1) && (gridPos.y == 0)
				|| (gridPos.x == par->NXC - 1) && (gridPos.y == par->NYC - 1))
			{
				int2 gridPos2;

				if (gridPos.x == 0) gridPos2.x = par->NXC - 1;
				else gridPos2.x = 0;

				gridPos2.y = gridPos.y;

				uint gridHash = calcGridHash(gridPos2, par);
				uint startIndex = cellStart[gridHash];

				if (startIndex != 0xffffffff)
				{
					uint endIndex = cellEnd[gridHash];
					for (uint j = startIndex; j < endIndex; j++)
					{
						real2 dpos, dvel;
						real2 pos1 = p[index].pos;
						if (gridPos.x == 0) pos1.x += par->XCV;
						else pos1.x -= par->XCV;
						real2 pos2 = pPDPF[j].pos;
						real2 vel1 = p[index].vel;
						real2 vel2 = pPDPF[j].vel;

						if (gridPos.y == par->NYC - 1)
						{
							calcRelPosVelNoSlip(pos2, pos1, vel2, vel1, 1, &dpos, &dvel, par);
							result += interaction(j, pPDPF, index, p, dpos, dvel, par);
						}
						if (gridPos.y == 0)
						{
							calcRelPosVelNoSlip(pos2, pos1, vel2, vel1, 3, &dpos, &dvel, par);
							result += interaction(j, pPDPF, index, p, dpos, dvel, par);
						}

					}
				}
			}

		}

		p[index].rh_vel.x = -( (result.x * p[index].p) + result.z) / p[index].d;
		p[index].rh_vel.y = -( (result.y * p[index].p) + result.u) / p[index].d;
		p[index].rh_m = -result.v / (pPDPF[0].di);
	}
}
