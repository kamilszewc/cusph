#include "hip/hip_runtime.h"
/*
* calcInteractionParticles.cu
*
*  Created on: 27-10-2015
*      Author: Kamil Szewc (kamil.szewc@gmail.com)
*/
#include "../../sph.h"
#include "../../hlp.h"
#include "../../methods/kernels.cuh"
#include "../../methods/calcGridHash.cuh"
#include "../../methods/calcGridPos.cuh"
#include "../../methods/calcRelPosVelNoSlip.cuh"

__device__ static float calcK(real2 dvel, real dLiquid, real oLiquid, real oDust, real d, real visc)
{
	real u = sqrt(pow2(dvel.x) + pow2(dvel.y));

	real re = d * u / visc;

	real cd = 0.0;
	if (re > 0.0) cd = 24.0 * sqrt(1.0 + (3.0 * re / 16.0)) / re;

	return (3.0 / 4.0) * dLiquid * oLiquid * oDust * cd * u * pow(oLiquid, -2.65) / d;
}

__device__ static real4 interaction(uint j, Particle *pFluid, uint i, Particle *pParticle, real2 dpos, real2 dvel, Parameters *par)
{
	real r = sqrt(pow2(dpos.x) + pow2(dpos.y));
	real q = r * par->I_H;

	if (q < 2.0)
	{
		real gkx = grad_of_kern(dpos.x, q, par->I_H);
		real gky = grad_of_kern(dpos.y, q, par->I_H);
		real k = kern_kwon_monaghan(q, par->I_H);

		real D = 0.0005;

		real K = calcK(dvel, pFluid[j].d, pFluid[j].o, pParticle[i].o, D, pFluid[j].nu);

		real pres = pFluid[j].m * pFluid[j].p / pFluid[j].d;

		real visc = 2.0 * pFluid[j].m * K * (dvel.x*dpos.x + dvel.y*dpos.y) * k / ((pow2(r) + 0.01*pow2(par->H)) * pFluid[j].d);


		return MAKE_REAL4(pres * gkx, pres * gky, visc * dpos.x, visc * dpos.y);
	}
	else
	{
		return MAKE_REAL4(0.0, 0.0, 0.0, 0.0);
	}
}



__global__ void calcInteractionFluidOnParticlesWSDP(Particle *p,
	uint *gridParticleIndex,
	uint *cellStart,
	uint *cellEnd,
	Particle *pPDPF,
	Parameters *par)
{
	uint index = threadIdx.x + blockIdx.x*blockDim.x;
	if (index < par->N_DISPERSED_PHASE_FLUID) {

		real2 pos = MAKE_REAL2(pPDPF[index].pos.x, pPDPF[index].pos.y);

		int2 gridPos = calcGridPos(pos, par);
		uint gridHash0 = calcGridHash(gridPos, par);
		real4 result = MAKE_REAL4(0.0, 0.0, 0.0, 0.0);

		for (int y = -1; y <= 1; y++) {
			for (int x = -1; x <= 1; x++) {
				int2 gridPos2;
				gridPos2.x = gridPos.x + x;
				gridPos2.y = gridPos.y + y;
				if ((gridPos2.x < 0) || (gridPos2.x > par->NXC - 1) || (gridPos2.y < 0) || (gridPos2.y > par->NYC - 1)) continue;

				uint gridHash = calcGridHash(gridPos2, par);
				uint startIndex = cellStart[gridHash];

				if (startIndex != 0xffffffff)
				{
					uint endIndex = cellEnd[gridHash];

					for (uint j = startIndex; j < endIndex; j++)
					{
						real2 dpos, dvel;
						real2 pos1 = pPDPF[index].pos;
						real2 pos2 = p[j].pos;
						real2 vel1 = pPDPF[index].vel;
						real2 vel2 = p[j].vel;
						calcRelPosVelNoSlip(pos1, pos2, vel1, vel2, 0, &dpos, &dvel, par);
						result += interaction(j, p, index, pPDPF, dpos, dvel, par);

						if (((gridPos.x == 0) && (gridPos2.x == 0)) || ((gridPos.x == par->NXC - 1) && (gridPos2.x == par->NXC - 1)) ||
							((gridPos.y == 0) && (gridPos2.y == 0)) || ((gridPos.y == par->NYC - 1) && (gridPos2.y == par->NYC - 1)))
						{
							if (par->T_BOUNDARY_PERIODICITY != 1)
							{
								if (gridPos.y == gridPos2.y)
								{
									if (gridPos.y == par->NYC - 1)
									{
										calcRelPosVelNoSlip(pos2, pos1, vel2, vel1, 1, &dpos, &dvel, par);
										result += interaction(j, p, index, pPDPF, dpos, dvel, par);
									}
									if (gridPos.y == 0)
									{
										calcRelPosVelNoSlip(pos2, pos1, vel2, vel1, 3, &dpos, &dvel, par);
										result += interaction(j, p, index, pPDPF, dpos, dvel, par);
									}
								}
							}
							if (par->T_BOUNDARY_PERIODICITY == 0)
							{
								if (gridPos.x == gridPos2.x)
								{
									if (gridPos2.x == 0)
									{
										calcRelPosVelNoSlip(pos2, pos1, vel2, vel1, 4, &dpos, &dvel, par);
										result += interaction(j, p, index, pPDPF, dpos, dvel, par);
									}
									if (gridPos2.x == par->NXC - 1)
									{
										calcRelPosVelNoSlip(pos2, pos1, vel2, vel1, 2, &dpos, &dvel, par);
										result += interaction(j, p, index, pPDPF, dpos, dvel, par);
									}
								}
								if ((gridPos.x == gridPos2.x) && (gridPos.y == gridPos.y))
								{
									if ((gridPos.x == 0) && (gridPos.y == 0))
									{
										calcRelPosVelNoSlip(pos2, pos1, vel2, vel1, 7, &dpos, &dvel, par);
										result += interaction(j, p, index, pPDPF, dpos, dvel, par);
									}
									if ((gridPos.x == par->NXC - 1) && (gridPos.y == 0))
									{
										calcRelPosVelNoSlip(pos2, pos1, vel2, vel1, 6, &dpos, &dvel, par);
										result += interaction(j, p, index, pPDPF, dpos, dvel, par);
									}
									if ((gridPos.x == 0) && (gridPos.y == par->NYC - 1))
									{
										calcRelPosVelNoSlip(pos2, pos1, vel2, vel1, 8, &dpos, &dvel, par);
										result += interaction(j, p, index, pPDPF, dpos, dvel, par);
									}
									if ((gridPos.x == par->NXC - 1) && (gridPos.y == par->NYC - 1))
									{
										calcRelPosVelNoSlip(pos2, pos1, vel2, vel1, 5, &dpos, &dvel, par);
										result += interaction(j, p, index, pPDPF, dpos, dvel, par);
									}
								}
							}

						}
					}

				}
			}
		}

		if ((par->T_BOUNDARY_PERIODICITY > 0) && ((gridPos.x == 0) || (gridPos.x == par->NXC - 1)))
		{
			for (int y = -1; y <= 1; y++)
			{
				int2 gridPos2;
				if (gridPos.x == 0) gridPos2.x = par->NXC - 1;
				if (gridPos.x == par->NXC - 1) gridPos2.x = 0;
				gridPos2.y = gridPos.y + y;
				if ((gridPos2.y < 0) || (gridPos2.y > par->NYC - 1)) continue;

				uint gridHash = calcGridHash(gridPos2, par);
				uint startIndex = cellStart[gridHash];

				if (startIndex != 0xffffffff)
				{
					uint endIndex = cellEnd[gridHash];

					for (uint j = startIndex; j < endIndex; j++)
					{
						real2 dpos, dvel;
						real2 pos1 = pPDPF[index].pos;
						if (gridPos.x == 0) pos1.x += par->XCV;
						if (gridPos.x == par->NXC - 1) pos1.x -= par->XCV;
						real2 pos2 = p[j].pos;
						real2 vel1 = pPDPF[index].vel;
						real2 vel2 = p[j].vel;

						calcRelPosVelNoSlip(pos1, pos2, vel1, vel2, 0, &dpos, &dvel, par);
						result += interaction(j, p, index, pPDPF, dpos, dvel, par);
					}

				}
			}
		}


		if ((par->T_BOUNDARY_PERIODICITY == 1) && ((gridPos.y == 0) || (gridPos.y == par->NYC - 1)))
		{
			for (int x = -1; x <= 1; x++)
			{
				int2 gridPos2;
				if (gridPos.y == 0) gridPos2.y = par->NYC - 1;
				if (gridPos.y == par->NYC - 1) gridPos2.y = 0;
				gridPos2.x = gridPos.x + x;
				if ((gridPos2.x < 0) || (gridPos2.x > par->NXC - 1)) continue;

				uint gridHash = calcGridHash(gridPos2, par);
				uint startIndex = cellStart[gridHash];

				if (startIndex != 0xffffffff)
				{
					uint endIndex = cellEnd[gridHash];

					for (uint j = startIndex; j < endIndex; j++)
					{
						real2 dpos, dvel;
						real2 pos1 = pPDPF[index].pos;
						if (gridPos.y == 0) pos1.y += par->YCV;
						if (gridPos.y == par->NYC - 1) pos1.y -= par->YCV;
						real2 pos2 = p[j].pos;
						real2 vel1 = pPDPF[index].vel;
						real2 vel2 = p[j].vel;

						calcRelPosVelNoSlip(pos1, pos2, vel1, vel2, 0, &dpos, &dvel, par);
						result += interaction(j, p, index, pPDPF, dpos, dvel, par);
					}

				}
			}
		}

		if (par->T_BOUNDARY_PERIODICITY == 1)
		{
			if (((gridPos.x == 0) && (gridPos.y == 0))
				|| (gridPos.x == 0) && (gridPos.y == par->NYC - 1)
				|| (gridPos.x == par->NXC - 1) && (gridPos.y == 0)
				|| (gridPos.x == par->NXC - 1) && (gridPos.y == par->NYC - 1))
			{
				int2 gridPos2;

				if (gridPos.x == 0) gridPos2.x = par->NXC - 1;
				else gridPos2.x = 0;

				if (gridPos.y == 0) gridPos2.y = par->NYC - 1;
				else gridPos2.y = 0;

				uint gridHash = calcGridHash(gridPos2, par);
				uint startIndex = cellStart[gridHash];

				if (startIndex != 0xffffffff)
				{
					uint endIndex = cellEnd[gridHash];
					for (uint j = startIndex; j < endIndex; j++)
					{
						real2 dpos, dvel;
						real2 pos1 = pPDPF[index].pos;
						if (gridPos.x == 0) pos1.x += par->XCV;
						else pos1.x -= par->XCV;
						if (gridPos.y == 0) pos1.y += par->YCV;
						else pos1.y -= par->YCV;
						real2 pos2 = p[j].pos;
						real2 vel1 = pPDPF[index].vel;
						real2 vel2 = p[j].vel;

						calcRelPosVelNoSlip(pos1, pos2, vel1, vel2, 0, &dpos, &dvel, par);
						result += interaction(j, p, index, pPDPF, dpos, dvel, par);
					}
				}
			}

		}

		if (par->T_BOUNDARY_PERIODICITY == 2)
		{
			if (((gridPos.x == 0) && (gridPos.y == 0))
				|| (gridPos.x == 0) && (gridPos.y == par->NYC - 1)
				|| (gridPos.x == par->NXC - 1) && (gridPos.y == 0)
				|| (gridPos.x == par->NXC - 1) && (gridPos.y == par->NYC - 1))
			{
				int2 gridPos2;

				if (gridPos.x == 0) gridPos2.x = par->NXC - 1;
				else gridPos2.x = 0;

				gridPos2.y = gridPos.y;

				uint gridHash = calcGridHash(gridPos2, par);
				uint startIndex = cellStart[gridHash];

				if (startIndex != 0xffffffff)
				{
					uint endIndex = cellEnd[gridHash];
					for (uint j = startIndex; j < endIndex; j++)
					{
						real2 dpos, dvel;
						real2 pos1 = pPDPF[index].pos;
						if (gridPos.x == 0) pos1.x += par->XCV;
						else pos1.x -= par->XCV;
						real2 pos2 = p[j].pos;
						real2 vel1 = pPDPF[index].vel;
						real2 vel2 = p[j].vel;

						if (gridPos.y == par->NYC - 1)
						{
							calcRelPosVelNoSlip(pos2, pos1, vel2, vel1, 1, &dpos, &dvel, par);
							result += interaction(j, p, index, pPDPF, dpos, dvel, par);
						}
						if (gridPos.y == 0)
						{
							calcRelPosVelNoSlip(pos2, pos1, vel2, vel1, 3, &dpos, &dvel, par);
							result += interaction(j, p, index, pPDPF, dpos, dvel, par);
						}

					}
				}
			}

		}

		pPDPF[index].rh_vel.x = - (pPDPF[index].o * result.x / pPDPF[index].d) - (result.z / pPDPF[index].d);
		pPDPF[index].rh_vel.y = - (pPDPF[index].o * result.y / pPDPF[index].d) - (result.w / pPDPF[index].d);

	}
}
