#include "hip/hip_runtime.h"
/*
 * calcHydrostaticPressure.cu
 *
 *  Created on: 28-05-2015
 *      Author: Kamil Szewc (kamil.szewc@gmail.com)
 */

#include "../../../sph.h"
#include "../../../hlp.h"
#include "../../../methods/kernels.cuh"
#include "../../../methods/interactions.cuh"

#include <stdio.h>


__device__ static real interaction(uint i, uint j, real2 dpos, real2 dvel, Particle *p, Parameters *par)
{
	real r = sqrt(pow2(dpos.x) + pow2(dpos.y));
	real q = r * par->I_H;

	if (q < 2.0)
	{
		real k = kern(q, par->I_H);

		return k*p[i].m/p[i].d;
	}
	else 
	{
		return 0.0;
	}
}


__global__ void calcHydrostaticPressure(Particle *p,
	uint *gridParticleIndex,
	uint *cellStart,
	uint *cellEnd,
	Parameters *par)
{
	uint index = threadIdx.x + blockIdx.x*blockDim.x;
	if (index < par->N) {
		
		p[index].ph = 0.0;
		real height = 0.0;

		real2 pos = MAKE_REAL2(p[index].pos.x, p[index].pos.y);

		int2 gridPos = calcGridPos(pos, par);

		//if (p[index].id == 20060) printf("--------------\n");
		for (int yc = par->NYC-1; yc >= gridPos.y; yc--)
		{
			int2 gridPosCheck;
			gridPosCheck.x = gridPos.x;
			gridPosCheck.y = yc;
			uint gridHash = calcGridHash(gridPosCheck, par);
			uint startIndex = cellStart[gridHash];
			
			if (startIndex == 0xffffffff) continue;

			for (int i=5; i>=0; i--)
			{
				real y = (real)yc*2.0*par->H + i*2.0*par->H/5.0;
				real result = 0.0;

				//if (p[index].id == 20060) printf("y=%f ", y);

				for (int iy = -1; iy <= 1; iy++) {
					for (int ix = -1; ix <= 1; ix++) {
						gridPosCheck.x = gridPos.x + ix;
						gridPosCheck.y = yc + iy;

						uint gridHash = calcGridHash(gridPosCheck, par);
						uint startIndex = cellStart[gridHash];

						if (startIndex != 0xffffffff)
						{
							uint endIndex = cellEnd[gridHash];

							for (uint j = startIndex; j < endIndex; j++)
							{
								real2 dpos, dvel;
								real2 pos1 = p[index].pos;
								pos1.y = y;
								real2 pos2 = p[j].pos;
								real2 vel1 = p[j].vel;
								real2 vel2 = p[j].vel;

								calcRelPosVelNoSlip(pos1, pos2, vel1, vel2, 0, &dpos, &dvel, par);
								result += interaction(index, j, dpos, dvel, p, par);
							}
						}
					}
				}

				//if (p[index].id == 20060 ) printf("result=%f\n", result);
				if (result >= 0.5)
				{
					height = y;
					break;
				}
			}

			if (height != 0.0) break;
		}

		if (height > 0.0)
		{
			p[index].ph = p[index].di * fabs(par->G_Y) * (height-p[index].pos.y);
		}
	}
}
