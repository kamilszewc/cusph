#include "hip/hip_runtime.h"
/*
 * smoothingDensity.cu
 *
 *  Created on: 25-08-2015
 *      Author: Kamil Szewc (kamil.szewc@gmail.com)
 *              Michal Olejnik
 */

#include "../../../sph.h"
#include "../../../hlp.h"
#include "../../../methods/kernels.cuh"
#include "../../../methods/interactions.cuh"

__device__ static real2 interaction(uint i, uint j, real2 dpos, real2 dvel, Particle *p, Parameters *par)
{
	real r = sqrt(pow2(dpos.x) + pow2(dpos.y));
	real q = r * par->I_H;

	if (q < 2.0)
	{
		real k = kern(q, par->I_H);
		return MAKE_REAL2( k , k * p[j].m / p[j].d);
	}
	else 
	{
		return MAKE_REAL2(0.0, 0.0);
	}
}


__global__ void smoothingDensity(Particle *p,
	uint *gridParticleIndex,
	uint *cellStart,
	uint *cellEnd,
	Parameters *par)
{
	uint index = threadIdx.x + blockIdx.x*blockDim.x;
	if (index < par->N) {
		register real2 result = MAKE_REAL2(0.0,0.0);
		#include "../../../methods/interactions/interactionsPositiveOnWallNoSlip.cuh"

		p[index].d = p[index].m * result.x / result.y;
	}
}
