#include "hip/hip_runtime.h"
/*
* calcPressureWHA.cu
*
*  Created on: 11-04-2013
*      Author: Kamil Szewc
*/
#include <math.h>
#include "../../sph.h"
#include "../../hlp.h"

__global__ void calcPressureWHA(Particle *p, Parameters *par)
{
	unsigned int tid = threadIdx.x + blockIdx.x*blockDim.x;
	while (tid < par->N) {
		p[tid].p = p[tid].b * (pow(p[tid].d / p[tid].di, p[tid].gamma) - 1.0);
		tid += blockDim.x * gridDim.x;
	}
}


