#include "hip/hip_runtime.h"
/*
* calcDensityWHA.cu
*
*  Created on: 02-09-2013
*      Author: Kamil Szewc (kamil.szewc@gmail.com)
*/
#include <math.h>
#include "../../sph.h"
#include "../../hlp.h"
#include "../../methods/kernels.cuh"
#include "../../methods/interactions.cuh"


__device__ static real interaction(uint i, uint j, real2 dpos, real2 dvel, Particle *p, Parameters *par)
{
	real q = sqrt(pow2(dpos.x) + pow2(dpos.y)) * par->I_H;
	if (q < 2.0) return kern(q, par->I_H);
	else return 0.0;
}


__global__ void calcDensityWHA(Particle *p,
	uint *gridParticleIndex,
	uint *cellStart,
	uint *cellEnd,
	Parameters *par)
{
	uint index = threadIdx.x + blockIdx.x*blockDim.x;
	if (index < par->N) {
		register real result = 0.0;
		#include "../../methods/interactions/interactionsPositiveOnWallNoSlip.cuh"

		p[index].d = result * p[index].m;
		p[index].o = pow2(1.0 / result);
	}
}
