#include "hip/hip_runtime.h"
/*
* calcAreaWHA.cu
*
*  Created on: 02-09-2013
*      Author: Kamil Szewc (kamil.szewc@gmail.com)
*/
#include <math.h>
#include "../../sph.h"
#include "../../hlp.h"
#include "../../methods/kernels.cuh"
#include "../../methods/interactions.cuh"


__global__ void calcAreaWHA(Particle *p,
	uint *gridParticleIndex,
	uint *cellStart,
	uint *cellEnd,
	Parameters *par)
{
	uint index = threadIdx.x + blockIdx.x*blockDim.x;
	if (index < par->N) {
		p[index].a = p[index].n.z * p[index].m / p[index].d;
	}
}